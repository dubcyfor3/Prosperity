#include "hip/hip_runtime.h"
#include "prosparsity_cuda.hpp"
#include <hip/hip_runtime.h>
#include <torch/torch.h>
#include <torch/extension.h>

#define TILE_SIZE_M 256
#define TILE_SIZE_K 16


__global__ void prosparsity_kernel(
    uint8_t *__restrict__ input_act,
    uint8_t *__restrict__ prosparsity_act,
    int *__restrict__ prefix_array,
    int M,
    int K
)
{

    int tile_m = blockIdx.x;
    int tile_k = blockIdx.y;

    int start_m = tile_m * TILE_SIZE_M;
    int start_k = tile_k * TILE_SIZE_K;

    int end_m = min(start_m + TILE_SIZE_M, M);
    int end_k = min(start_k + TILE_SIZE_K, K);

    int m = start_m + threadIdx.x;

    extern __shared__ char shared_mem[];

    int* nnz_array = (int*)shared_mem;
    uint8_t* act_tile = (uint8_t*)(shared_mem + TILE_SIZE_M * sizeof(int));

    nnz_array[threadIdx.x] = 0;
    // copy the input act to shared memory
    if (m < end_m)
    {
        for (int k = start_k; k < end_k; k++)
        {
            act_tile[threadIdx.x * TILE_SIZE_K + k - start_k] = input_act[m * K + k];
            nnz_array[threadIdx.x] += input_act[m * K + k];
        }
    }

    __syncthreads();
    bool is_subset;
    int max_subset = 0;
    int prefix = -1;

    if (m < end_m)
    {
        for (int i = start_m; i < end_m; i++)
        {
            is_subset = !(nnz_array[threadIdx.x] == nnz_array[i - start_m] && threadIdx.x <= i - start_m);
            for (int k = start_k; k < end_k; k++)
            {
                // is_subset &= input_act[m * K + k] >= input_act[i * K + k];
                is_subset &= act_tile[threadIdx.x * TILE_SIZE_K + k - start_k] >= act_tile[(i - start_m) * TILE_SIZE_K + k - start_k];
            }
            if (is_subset && nnz_array[i - start_m] > max_subset)
            {
                max_subset = nnz_array[i - start_m];
                prefix = i - start_m;
            }
        }
        
        if (nnz_array[threadIdx.x] < 2)
        {
            prefix = -1;
        }

        prefix_array[tile_m * gridDim.y * TILE_SIZE_M + tile_k * TILE_SIZE_M + m - start_m] = prefix;
        if (prefix != -1)
        {
            for (int k = start_k; k < end_k; k++)
            {
                // prosparsity_act[m * K + k] = input_act[m * K + k] - input_act[(prefix + start_m) * K + k];
                prosparsity_act[m * K + k] = act_tile[threadIdx.x * TILE_SIZE_K + k - start_k] - act_tile[prefix * TILE_SIZE_K + k - start_k];
            }
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor> find_product_sparsity(
    torch::Tensor _input_act
)
{
    int M = _input_act.size(0);
    int K = _input_act.size(1);
    int num_tiles_m = (M + TILE_SIZE_M - 1) / TILE_SIZE_M;
    int num_tiles_k = (K + TILE_SIZE_K - 1) / TILE_SIZE_K;

    // convert input act from bool to uint8_t
    _input_act = _input_act.to(torch::kByte);

    // move to device
    _input_act = _input_act.to(torch::kCUDA);



    auto input_act = reinterpret_cast<uint8_t*>(_input_act.data_ptr<uint8_t>());

    // clone the input_act tensor to prosparsity_act
    at::Tensor _prosparsity_act = _input_act.clone();


    at::Tensor _prefix_array = torch::ones({num_tiles_m, num_tiles_k, TILE_SIZE_M}, torch::kInt32).to(torch::kCUDA);

    _prefix_array = _prefix_array * -1;


    auto prosparsity_act = reinterpret_cast<uint8_t*>(_prosparsity_act.data_ptr<uint8_t>());
    auto prefix_array = reinterpret_cast<int*>(_prefix_array.data_ptr<int>());

    dim3 num_blocks(num_tiles_m, num_tiles_k);
    dim3 threads_per_block(TILE_SIZE_M);
    int shared_memory_size = TILE_SIZE_M * sizeof(int) + TILE_SIZE_M * TILE_SIZE_K * sizeof(uint8_t);

    if (shared_memory_size > 49152)
    {
        printf("Error: too much shared memory required\n");
        // move to CPU
        _prosparsity_act = _prosparsity_act.to(torch::kCPU);
        _prefix_array = _prefix_array.to(torch::kCPU);
        return std::make_tuple(_prosparsity_act, _prefix_array);
    }



    prosparsity_kernel<<<num_blocks, threads_per_block, shared_memory_size>>>(
    // prosparsity_kernel<<<num_blocks, threads_per_block>>>(
        input_act,
        prosparsity_act,
        prefix_array,
        M,
        K
    );

    // input_act = input_act.to(torch::kCPU);
    _prosparsity_act = _prosparsity_act.to(torch::kCPU);
    _prefix_array = _prefix_array.to(torch::kCPU);

    return std::make_tuple(_prosparsity_act, _prefix_array);
}

void find_product_sparsity_cpp(
    uint8_t* input_act,
    uint8_t* prosparsity_act,
    int* prefix_array,
    int M,
    int K
)
{
    int num_tiles_m = (M + TILE_SIZE_M - 1) / TILE_SIZE_M;
    int num_tiles_k = (K + TILE_SIZE_K - 1) / TILE_SIZE_K;

    // Allocate memory for the input_act on the device
    uint8_t* d_input_act;
    hipMalloc(&d_input_act, M * K * sizeof(uint8_t));

    // Copy the input_act from the host (CPU) to the device (GPU)
    hipMemcpy(d_input_act, input_act, M * K * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Allocate memory for the output prosparsity_act on the device
    uint8_t* d_prosparsity_act;
    hipMalloc(&d_prosparsity_act, M * K * sizeof(uint8_t));

    // copy d_input_act to d_prosparsity_act
    hipMemcpy(d_prosparsity_act, d_input_act, M * K * sizeof(uint8_t), hipMemcpyDeviceToDevice);

    // Allocate memory for the prefix_array on the device
    int* d_prefix_array;
    hipMalloc(&d_prefix_array, num_tiles_m * num_tiles_k * TILE_SIZE_M * sizeof(int));

    // Set prefix array to -1 on the device
    hipMemset(d_prefix_array, -1, num_tiles_m * num_tiles_k * TILE_SIZE_M * sizeof(int));

    // Configure the kernel execution parameters
    dim3 num_blocks(num_tiles_m, num_tiles_k);
    dim3 threads_per_block(TILE_SIZE_M);
    int shared_memory_size = TILE_SIZE_M * sizeof(int) + TILE_SIZE_M * TILE_SIZE_K * sizeof(uint8_t);

    if (shared_memory_size > 49152)
    {
        printf("Error: too much shared memory required\n");
        hipFree(d_input_act);
        hipFree(d_prosparsity_act);
        hipFree(d_prefix_array);
        return;
    }

    // Launch the kernel
    prosparsity_kernel<<<num_blocks, threads_per_block, shared_memory_size>>>(
        d_input_act,
        d_prosparsity_act,
        d_prefix_array,
        M,
        K
    );

    // Check for errors during kernel execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        hipFree(d_input_act);
        hipFree(d_prosparsity_act);
        hipFree(d_prefix_array);
        return;
    }

    // move the prosparsity_act array from the GPU to the CPU

    hipMemcpy(prosparsity_act, d_prosparsity_act, M * K * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipMemcpy(prefix_array, d_prefix_array, num_tiles_m * num_tiles_k * TILE_SIZE_M * sizeof(int), hipMemcpyDeviceToHost);

    // Free the input and prefix array memory on the GPU as they are no longer needed
    hipFree(d_input_act);
    hipFree(d_prefix_array);
    hipFree(d_prosparsity_act);

    return;
}
